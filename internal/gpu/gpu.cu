#include <stdio.h>
#include <hip/hip_runtime.h>

// Device (kernel)
__global__ void square_cuda(float *a, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N)
    a[idx] = a[idx] * a[idx];
}

extern "C"
{
  // Host driver - implements gpu.h
  void square(float *a, int N)
  {
    float *a_d;
    size_t size = N * sizeof(float);

    // Allocate memory on the GPU
    hipMalloc((void **)&a_d, size);

    // Copy the input data from CPU memory to GPU memory
    hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);

    // Launch the GPU kernel to do work
    int block_size = 4;
    int n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
    square_cuda<<<n_blocks, block_size>>>(a_d, N);

    // Copy the result data from GPU memory back to our CPU memory
    hipMemcpy(a, a_d, size, hipMemcpyDeviceToHost);

    // Free the GPU memory
    hipFree(a_d);
  }
}
