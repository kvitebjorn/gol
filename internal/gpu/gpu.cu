#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "cudart_loader.h"

// Device (kernel)
__global__ void square_cuda(float *a, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N)
    a[idx] = a[idx] * a[idx];
}

// Device `tick` (kernel). src and dst are flat row-major int arrays (0 or 1).
__global__ void tick_cuda(const int *src, int *dst, int rows, int cols)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int total = rows * cols;
  if (idx >= total)
    return;

  int r = idx / cols;
  int c = idx % cols;

  int alive = 0;

  // iterate neighbors
  for (int dr = -1; dr <= 1; ++dr)
  {
    for (int dc = -1; dc <= 1; ++dc)
    {
      if (dr == 0 && dc == 0)
        continue;
      int rr = r + dr;
      int cc = c + dc;

      // bounds check: treat outside as dead
      if (rr < 0 || rr >= rows || cc < 0 || cc >= cols)
        continue;
      int nidx = rr * cols + cc;
      alive += src[nidx];
    }
  }

  int cur = src[idx];
  int next = 0;
  if (cur == 1)
  {
    // live cell: survives if 2 or 3 neighbors
    next = (alive == 2 || alive == 3) ? 1 : 0;
  }
  else
  {
    // dead cell: becomes alive if exactly 3 neighbors
    next = (alive == 3) ? 1 : 0;
  }
  dst[idx] = next;
}

extern "C"
{
  // Host driver - implements gpu.h `square`
  void square(float *a, int N)
  {
    float *a_d;
    size_t size = N * sizeof(float);

    // Allocate memory on the GPU
    cudaMalloc_wrap((void **)&a_d, size);

    // Copy the input data from CPU memory to GPU memory
    cudaMemcpy_wrap(a_d, a, size, hipMemcpyHostToDevice);

    // Launch the GPU kernel to do work
    int block_size = 4;
    int n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
    square_cuda<<<n_blocks, block_size>>>(a_d, N);

    // Copy the result data from GPU memory back to our CPU memory
    cudaMemcpy_wrap(a, a_d, size, hipMemcpyDeviceToHost);

    // Free the GPU memory
    cudaFree_wrap(a_d);
  }

  // Host driver - implements gpu.h `tick`
  void tick(int *src, int *dst, int rows, int cols)
  {
    size_t n = (size_t)rows * (size_t)cols;
    if (n == 0)
      return;

    int *src_d = nullptr;
    int *dst_d = nullptr;
    size_t bytes = n * sizeof(int);

    hipError_t err;

    // Allocate memory on the device for src & dst
    err = cudaMalloc_wrap((void **)&src_d, bytes);
    if (err != hipSuccess)
    {
      fprintf(stderr, "hipMalloc src failed: %s\n", cudaGetErrorString_wrap(err));
      return;
    }
    err = cudaMalloc_wrap((void **)&dst_d, bytes);
    if (err != hipSuccess)
    {
      fprintf(stderr, "hipMalloc dst failed: %s\n", cudaGetErrorString_wrap(err));
      cudaFree_wrap(src_d);
      return;
    }

    // Copy values to the device for src
    err = cudaMemcpy_wrap(src_d, src, bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
      fprintf(stderr, "hipMemcpy to device failed: %s\n", cudaGetErrorString_wrap(err));
      cudaFree_wrap(src_d);
      cudaFree_wrap(dst_d);
      return;
    }

    // Launch the kernel to do the thing
    int block_size = 256;
    int n_blocks = (int)((n + block_size - 1) / block_size);

    tick_cuda<<<n_blocks, block_size>>>(src_d, dst_d, rows, cols);
    cudaDeviceSynchronize_wrap();

    // Copy the device dst to our host
    err = cudaMemcpy_wrap(dst, dst_d, bytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
      fprintf(stderr, "hipMemcpy to host failed: %s\n", cudaGetErrorString_wrap(err));
    }

    cudaFree_wrap(src_d);
    cudaFree_wrap(dst_d);
  }
}
